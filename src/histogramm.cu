#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES

#include <cstdio>
#include <cstring>
#include <memory>
#include "../include/renderer.h"
#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"

static constexpr int num_threads = 256;

__global__ void histo_kernel(unsigned char* buffer,
                     const size_t size, unsigned int* histo)
{
    __shared__ unsigned int cache[num_threads];
    cache[threadIdx.x] = 0u;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int offset = blockDim.x * gridDim.x;
  
    while (i < size) {
        atomicAdd(&(cache[buffer[i]]), 1u);
        i += offset;
    }
    __syncthreads();
    atomicAdd(&(histo[threadIdx.x]), cache[threadIdx.x]);
}

size_t readFromFile(unsigned char**, const char*);

template <typename T>
static void getBufferData(const GLuint Buf, T* buffer, const size_t size);

int main(void)
{
    unsigned char* buffer = nullptr;
    const size_t size = readFromFile(&buffer, "../resource/example.xml");
    
    hipDeviceProp_t prop;
    int dev;
    std::memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 0;
    hipChooseDevice(&dev, &prop);
    cudaGLSetGLDevice(dev);
    //hipSetDevice(0);
    hipGetDeviceProperties(&prop, dev);
    const int blocks = prop.multiProcessorCount;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    unsigned char* dev_buffer;
    hipMalloc((void**)&dev_buffer, size);
    hipMemcpy(dev_buffer, buffer, size, hipMemcpyHostToDevice);
    hipHostFree(buffer);

    std::unique_ptr<GraphicsData> gdata = std::make_unique<GraphicsData>();
    initGL(*gdata, size);
    unsigned int* dev_histo = nullptr;
    hipGraphicsResource *resource;
    hipGraphicsGLRegisterBuffer(&resource, gdata->VBO, cudaGraphicsMapFlagsNone);
    hipGraphicsMapResources(1, &resource, NULL);
    size_t b_size;
    hipGraphicsResourceGetMappedPointer((void**)&dev_histo, &b_size, resource);

    histo_kernel<<<blocks*2, num_threads>>>(dev_buffer, size, dev_histo);

    hipFree(dev_buffer);
    hipGraphicsUnmapResources(1, &resource, NULL);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to calculate: %3.1f ms\n", elapsedTime);

    unsigned int* histo;
    histo  = (unsigned int*)malloc(sizeof(unsigned int)* buf_size);
    //hipMemcpy(histo, dev_histo, sizeof(unsigned int)*buf_size, 
   //                                 hipMemcpyDeviceToHost);
    getBufferData(gdata->VBO, histo, buf_size);
    puts("Histogramm data:");
    for (auto i = 0u; i < buf_size; i++)
        printf("%d\t", histo[i]);
    puts("--------------");
    free(histo);

    renderGL(*gdata);

    hipGraphicsUnregisterResource(resource);
    cleanupGL(*gdata);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

size_t readFromFile(unsigned char** buff, const char* filename)
{
    FILE* file = fopen(filename, "rb");
    if (file == nullptr) {
        puts("Error opening file");
    }
    size_t sz = 0u;
    fseek(file, 0l, SEEK_END);
    sz = ftell(file);
    rewind(file);

    hipHostAlloc((void**)buff, sz, hipHostMallocDefault);
    //hipDeviceSynchronize();
   
    constexpr auto stride = 16u;  
    auto sz1 = sz & (stride - 1u);
    auto sz0 = sz - sz1;

    alignas(stride) char symbol[stride];
    for (auto i = 0lu; i < sz0; i += stride) {
        fread(symbol, 1, stride, file);
        std::memcpy(((*buff)+i), symbol,  stride);
    }
    fread(symbol, 1, sz1, file);
    std::memcpy(((*buff)+sz0), symbol,  sz1);

    fclose(file);

    return sz;
}

template <typename T>
static void getBufferData(const GLuint buf, T* buffer, const size_t size)
{
    glBindBuffer(GL_ARRAY_BUFFER, buf);
    T* device_data = (T*)glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);
    memcpy(buffer, device_data, size * sizeof (T));
    glUnmapBuffer(GL_ARRAY_BUFFER);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
}
